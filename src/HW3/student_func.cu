#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Definition Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
#include <cfloat>

template <typename T>
struct IBinary_Operator
{
public:
    __device__ __host__ virtual T operator() (const T& a, const T& b)=0;
};

template <typename T>
struct Max_Operator: public IBinary_Operator<T>
{
public:
    __device__ __host__ virtual T operator() (const T&a, const T& b){
        T result;
        if(isnan(a))
            return b;
        if(isnan(b))
            return a;
        result =  (a<b)? b : a;
        return result;
    }
};

template <typename T>
struct Min_Operator: public IBinary_Operator<T>
{
public:
    __device__ __host__ virtual T operator() (const T&a, const T& b){
        T result;
        if(isnan(a))
            return b;
        if(isnan(b))
            return a;
        result =  (a>b)? b : a;
        return result;
    }
};

template <typename T, typename T_Bin_Op>
__global__ void _reduction_global_mem_sub_( T * d_out,
                                            T *  d_in,
                                            const size_t totalCount,
                                            T_Bin_Op operation)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    if(myId >= totalCount)
        return;
    // do reduction in global memory

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if(myId +s < totalCount)
                d_in[myId] = operation(d_in[myId], d_in[myId + s]);
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global memory
    if (tid == 0)
        d_out[blockIdx.x] = d_in[myId];


}
/*wrapper function for global memory*/
template <typename T, typename T_Bin_Op>
void reduction_global_mem(  const T* const d_in,
                            const size_t numElement,
                            T* d_out,
                            T_Bin_Op operation)
{
    /*Two passes reduction*/

    /*problem scale*/
    const int maxThreadsPerBlock = 1024;
    int threads = maxThreadsPerBlock;
    int blocks = (numElement - 1)/ maxThreadsPerBlock + 1;

    /*clone d_in --> d_input_array*/
    T* d_input_array;

    checkCudaErrors(hipMalloc(&d_input_array,    sizeof(T) * numElement));
    checkCudaErrors(hipMemcpy(d_input_array,   d_in,   sizeof(T) * numElement, hipMemcpyDeviceToDevice));

    /*allocate intermediate result for first pass*/
    T* d_intermediate_result;
    checkCudaErrors(hipMalloc(&d_intermediate_result,    sizeof(T) * blocks));

    /*On first pass, compute local reduction per each thread block*/
    _reduction_global_mem_sub_<T,T_Bin_Op><<<blocks, threads>>>(d_intermediate_result, d_input_array, numElement, operation);

    /*On second pass, compute global reduction*/
    _reduction_global_mem_sub_<T,T_Bin_Op><<<1, threads>>>(d_out, d_intermediate_result, blocks, operation);

    checkCudaErrors(hipFree(d_input_array));

}

template <typename T, typename T_Bin_Op>
__global__ void _reduction_shared_mem_sub_( T * d_out,
                                            const T * const d_in,
                                            const size_t totalCount,
                                            T_Bin_Op operation)
{
    /*halve the number of threads in one block*/
    int myId = threadIdx.x + 2 * blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    if(myId >= totalCount)
        return;

    extern __shared__ T s_array[];
    /*reduce directly in first step*/
    int gap = blockDim.x;
    if(myId + gap < totalCount)
        s_array[tid] = operation(d_in[myId], d_in[myId + gap]);
    else
        s_array[tid] = d_in[myId];

    /*synch all threads within block*/
    __syncthreads();


    // do reduction in shared memory

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if(myId +s < totalCount)
                s_array[tid] = operation(s_array[tid], s_array[tid + s]);
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global memory
    if (tid == 0)
        d_out[blockIdx.x] = s_array[0];


}

/*wrapper function for shared memory*/
template <typename T, typename T_Bin_Op>
void reduction_shared_mem(  const T* const d_in,
                            const size_t numElement,
                            T* d_out,
                            T_Bin_Op operation)
{
    /*Two passes reduction*/

    /*problem scale*/
    const int maxThreadsPerBlock = 1024;
    int threads = maxThreadsPerBlock;
    int blocks = (numElement - 1)/ maxThreadsPerBlock + 1;
    T* d_intermediate_result;
    T* d_input_array = (T*) d_in;
    size_t input_array_element = numElement;

    do{
        /*allocate intermediate result for first pass*/
        checkCudaErrors(hipMalloc(&d_intermediate_result,    sizeof(T) * blocks));

        /*On first pass, compute local reduction per each thread block*/
        _reduction_shared_mem_sub_<T, T_Bin_Op><<<blocks, threads / 2, sizeof(T) * threads / 2>>>(d_intermediate_result, d_input_array, input_array_element, operation);

        if(d_input_array != d_in)
            checkCudaErrors(hipFree(d_input_array));

        /*On second pass, compute global reduction.
         *If the number of element of intermediate result can fit into one thread block, run final reduction.*/
        if(blocks <= maxThreadsPerBlock)

            _reduction_shared_mem_sub_<T, T_Bin_Op><<<1, threads / 2, sizeof(T) * threads / 2>>>(d_out, d_intermediate_result, blocks, operation);

        else{
            /*Otherwise, repeat first pass until it fits in one thread block.*/
            input_array_element = blocks;

            blocks = (blocks - 1)/ maxThreadsPerBlock + 1;

            d_input_array = d_intermediate_result;

        }

    }while(blocks > maxThreadsPerBlock);
}

template <typename T, typename T_Bin_Op>
void reduction( const T* const d_in,
                const size_t numElement,
                T * h_out,
                T_Bin_Op operation)

{
    /*allocate d_out*/
    T * d_out;

    checkCudaErrors(hipMalloc(&d_out,    sizeof(T) * 1));

    //reduction_global_mem<T, T_Bin_Op>(d_in, numElement, d_out, operation);

    reduction_shared_mem<T, T_Bin_Op>(d_in, numElement, d_out, operation);

    /*copy device output to host*/
    checkCudaErrors(hipMemcpy(h_out,   d_out,   sizeof(T) * 1, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_out));
}

template <typename T>
__global__ void _histogram_atomic_version_(	const T* const d_in,
											int* d_out,
											T min_logLum,
											T lumRange,
											const size_t numElement,
											const size_t numBins)
{
	int tid = threadIdx.x;

	int myId = blockDim.x * blockIdx.x + tid;

	if(myId > numElement)
		return;

	int binIndex = min( (int)floor((d_in[myId] - min_logLum) / lumRange * numBins), (int)numBins - 1 );

	atomicAdd(&(d_out[binIndex]), 1);
}

template <typename T>
void histogram( const T* const d_in,
				size_t numElements,
                size_t numBins,
                T min_logLum,
                T max_logLum,
                int* d_out)
{

    int threads = 512;

    int blocks = (numElements - 1) / threads + 1;

    float lumRange = max_logLum - min_logLum;

    _histogram_atomic_version_<float> <<<blocks, threads>>> (d_in, d_out, min_logLum, lumRange, numElements, numBins);

}
void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

	int * d_hist;
	/*allocate histogram output in device memory*/
	checkCudaErrors(hipMalloc(&d_hist, sizeof(int) * numBins));
	checkCudaErrors(hipMemset(d_hist, 0, sizeof(int) * numBins));

    reduction< float,Min_Operator<float> >(d_logLuminance, numRows * numCols, &min_logLum, Min_Operator<float>());

    reduction< float,Max_Operator<float> >(d_logLuminance, numRows * numCols, &max_logLum, Max_Operator<float>());

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    histogram< float > (d_logLuminance, numRows * numCols, numBins, min_logLum, max_logLum, d_hist);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    //debug
    int* h_hist;
    h_hist = (int*) malloc(sizeof(int) * numBins);
    checkCudaErrors(hipMemcpy(h_hist,   d_hist,   sizeof(int) * numBins, hipMemcpyDeviceToHost));
    for(int i=0;i<numBins;i++)
    	std::cout<<h_hist[i]<<" ";


    //exclusive_scan...
    checkCudaErrors(hipFree(d_hist));
}
